#include "hip/hip_runtime.h"

// TODO: Use 64 bit floats, when available in auro

import auro.system {
  void println (string);
  void error (string);
  void exit (int);
  int argc ();
  string argv (int);
}

import auro.buffer { type buffer; }

import auro.string {
  string itos (int);
  
  int length (string) as strlen;
  int codeof (char);
  char, int charat(string, int);
  string add (string, char) as addch;
  char newchar (int);
  string slice (string, int, int);

  string `new` (buffer) as newstr;
  buffer tobuffer (string);
}

import auro.math {
  float floor (float);
  float trunc (float);
  float mod (float, float) as fmod;
  float pow (float base, float exponent) as fpow;
}

any anyInt (int x) { return x as any; }
any anyStr (string x) { return x as any; }
any anyTable (Table x) { return x as any; }
any anyFn (Function x) { return x as any; }
any anyBool (bool x) { return x as any; }

bool testInt (any a) { return a is int; }
bool testStr (any a) { return a is string; }
bool testBool (any a) { return a is bool; }
bool testTable (any a) { return a is Table; }
bool testFn (any a) { return a is Function; }
bool testNil (any a) { return a is nil_t; }

int getInt (any a) { return a as int; }
string getStr (any a) { return a as string; }
bool getBool (any a) { return a as bool; }
Table getTable (any a) { return a as Table; }
Function getFn (any a) { return a as Function; }

import auro.utils.arraylist (any) {
  type `` as Array {
    any get (int);
    void push (any);
    void set (int, any);
    int len ();
  }
  Array `new` () as newArray;
}

import auro.function (Stack as in0, Stack as out0) {
  type `` as Function {
    Stack apply (Stack);
  }
  module `new` as newfn;
  module closure;
}

export Function;
export closure;

struct unit_t {bool dummy;}
type nil_t (unit_t);

struct Stack {
  int pos;
  Array arr;

  any first (Stack this) {
    if (this.more())
      return this.arr[this.pos];
    else return nil();
  }

  any next (Stack this) {
    if (this.more()) {
      any a = this.arr[this.pos];
      this.pos = this.pos + 1;
      return a;
    } else return nil();
  }

  void push (Stack this, any a) {
    this.arr.push(a);
  }

  bool more (Stack this) {
    return this.pos < this.arr.len();
  }

  void append (Stack this, Stack that) {
    int i = that.pos;
    while (i < that.arr.len()) {
      this.push(that.arr[i]);
      i = i+1;
    }
  }

  int length (Stack this) {
    int l = this.arr.len() - this.pos;
    if (l < 0) return 0;
    return l;
  }

  Stack copy (Stack this) { return new Stack(this.pos, this.arr); }
}

Stack newStack () {
  return new Stack(0, newArray());
}

bool isSpace (char ch) {
  int code = codeof(ch); // \t \n ' '
  return (code == 9) || (code == 10) || (code == 32);
}

any parseNum (string s) {
  int len = strlen(s);
  int i = 0;
  char ch;

  int value = 0;
  int anydigit = false;

  // Skip whitespace, fail or start hex
  while (i < len) {
    ch, i = charat(s, i);
    if ((codeof(ch) == 48) && (i < len)) { // '0'
      ch, i = charat(s, i);
      if ((codeof(ch) == 88) || (codeof(ch) == 120)) // Xx
        goto hexadecimal;
    }
    if (!isSpace(ch)) goto digit;
  }

  hexadecimal:
  while (i < len) {
    ch, i = charat(s, i);
    int c = codeof(ch);
    //if (codeof(ch) == 46) goto decimals;
    if ((c <= 57) && (c >= 48)) { // 0-9
      value = (value*16) + (c-48);
    } else if ((c <= 70) && (c >= 65)) { // A-F
      value = (value*16) + (c-65) + 10;
    } else if ((c >= 97) && (c <= 102)) { // a-f
      value = (value*16) + (c-97) + 10;
    } else goto endint;
  }

  // Decimal digits
  while (i < len) {
    ch, i = charat(s, i);
    digit:
    int c = codeof(ch);
    if ((c == 46) || (c == 69) || (c == 101)) // '.' 'E' 'e'
      goto floatPart;
    if ((c > 57) || (c < 48)) // not a digit
      goto endint; // break
    value = (value*10) + (c-48);
    anydigit = true;
  }

  endint:

  // Skip whitespace or fail
  while (i < len) {
    ch, i = charat(s, i);
    if (!isSpace(ch)) return nil();
  }

  return value as any;

  floatPart:

  float fval;
  float ten = itof(10);

  if (codeof(ch) == 46) { // '.'
    int digits = 0;

    // Decimal fractional digits
    while (i < len) {
      ch, i = charat(s, i);
      if ((codeof(ch) > 57) || (codeof(ch) < 48))
        goto endflt; // break
      value = (value*10) + (codeof(ch)-48);
      digits = digits + 1;
      anydigit = true;
    }
    endflt:

    fval = itof(value);
    while (digits > 0) {
      fval = fval / ten;
      digits = digits - 1;
    }
  } else fval = itof(value);

  if ((codeof(ch) == 69) || (codeof(ch) == 101)) { // Ee

    bool positive = true;
    int expval = 0;

    ch, i = charat(s, i); // skip 'e'

    if (codeof(ch) == 45) { // '-'
      ch, i = charat(s, i); // advance
      positive = false;
    } else if (codeof(ch) == 43) { // '+'
      ch, i = charat(s, i); // also advance
    }

    // At least 1 digit is required
    if ((codeof(ch) > 57) || (codeof(ch) < 48)) return nil();
    goto expdigit;

    while (i < len) {
      ch, i = charat(s, i);
      expdigit:
      if ((codeof(ch) > 57) || (codeof(ch) < 48))
        goto endexp; // break
      expval = (expval*10) + (codeof(ch)-48);
    }
    endexp:

    if (positive) {
      while (expval > 0) {
        fval = fval * ten;
        expval = expval - 1;
      }
    } else {
      while (expval > 0) {
        fval = fval / ten;
        expval = expval - 1;
      }
    }
  }

  if (!anydigit) return nil();
  
  while (i < len) {
    ch, i = charat(s, i);
    if (!isSpace(ch)) return nil();
  }

  return fval as any;
}

any getNum (any a) {
  if ((a is int) || (a is float)) return a;
  if (a is string) return parseNum(a as string);
  return nil();
}

float, float, bool getFloats (any a, any b) {
  float fa, fb;

  if (a is float) fa = a as float;
  else if (a is int) fa = itof(a as int);
  else return itof(0), itof(0), false;

  if (b is float) fb = b as float;
  else if (b is int) fb = itof(b as int);
  else return itof(0), itof(0), false;

  return fa, fb, true;
}

private any? meta_binop (any a, any b, string key) {
  Table? meta = get_metatable(a);
  if (meta.isnull()) {
    meta = get_metatable(b);
    if (meta.isnull()) goto err;
  }

  any index = meta.get().get(key as any);
  if (testFn(index)) {
    Function f = getFn(index);
    Stack args = newStack();
    args.push(a);
    args.push(b);
    Stack result = f.apply(args);
    return result.first() as any?;
  }

  err:
  return new any?();
}

private any meta_arith (any a, any b, string key) {
  any? r = meta_binop(a, b, key);
  if (r.isnull())
    error("Lua: attempt to perform arithmetic on a " + typestr(a) + " value");
  return r.get();
}

any add (any a, any b) {
  if ((a is int) && (b is int))
    return ((a as int) + (b as int)) as any;
  float fa, fb; bool t;
  fa, fb, t = getFloats(a, b);
  if (t) return (fa + fb) as any;
  return meta_arith(a, b, "__add");
}

any sub (any a, any b) {
  if ((a is int) && (b is int))
    return ((a as int) - (b as int)) as any;
  float fa, fb; bool t;
  fa, fb, t = getFloats(a, b);
  if (t) return (fa - fb) as any;
  return meta_arith(a, b, "__sub");
}

any mul (any a, any b) {
  if ((a is int) && (b is int))
    return ((a as int) * (b as int)) as any;
  float fa, fb; bool t;
  fa, fb, t = getFloats(a, b);
  if (t) return (fa * fb) as any;
  return meta_arith(a, b, "__mul");
}

any div (any a, any b) {
  float fa, fb; bool t;
  fa, fb, t = getFloats(a, b);
  if (t) return (fa / fb) as any;
  return meta_arith(a, b, "__div");
}

any idiv (any a, any b) {
  if ((a is int) && (b is int)) {
    int ia = a as int, ib = b as int;
    int r = ia / ib;
    // Int division truncates, but lua division floors
    // negative results are rounded the wrong way
    if ((r > 0) || ((ia * ib) > 0) || ((r * ib) == ia))
      return r as any;
    return (r - 1) as any;
  }
  float fa, fb; bool t;
  fa, fb, t = getFloats(a, b);
  if (t) return floor(fa / fb) as any;
  return meta_arith(a, b, "__idiv");
}

import auro.int {
  int mod (int, int) as imod;
}

any mod (any a, any b) {
  if ((a is int) && (b is int)) {
    int ia = a as int, ib = b as int;
    int r = imod(ia, ib);
    if ((r * ib) >= 0) return r as any;
    return (r + ib) as any;
  }
  float fa, fb; bool t;
  fa, fb, t = getFloats(a, b);
  if (!t) return meta_arith(a, b, "__mod");
  float r = fmod(fa, fb);
  if ((r * fb) >= itof(0)) return r as any;
  return (r + fb) as any;
}

any pow (any a, any b) {
  float fa, fb; bool t;
  fa, fb, t = getFloats(a, b);
  if (t) return fpow(fa, fb) as any;
  return meta_arith(a, b, "__pow");
}

any concat (any a, any b) {
  return anyStr(tostr(a) + tostr(b));
}

// TODO: Real nil, not just 0
any nil () { return (new unit_t(true) as nil_t) as any; }
any `true` () { return true as any; }
any `false` () { return false as any; }

export anyStr as string;
export anyInt as int;
export anyFn as function;

string typestr (any a) {
  if (testTable(a)) return "table";
  else if (testStr(a)) return "string";
  else if ((a is float) || testInt(a)) return "number";
  else if (testNil(a)) return "nil";
  else if (testBool(a)) return "bool";
  else if (testFn(a)) return "function";
  else return "userdata";
}

string tostr (any a) {
  if (testStr(a)) return getStr(a);
  else if (a is float) return ftos(a as float);
  else if (testInt(a)) return itos(getInt(a));
  else if (testBool(a)) {
    if (getBool(a))
      return "true";
    else
      return "false";
  }

  Table? meta = get_metatable(a);
  if (!meta.isnull()) {
    any index = meta.get().get("__tostring" as any);
    if (testFn(index)) {
      Function f = getFn(index);
      Stack args = newStack();
      args.push(a);
      Stack result = f.apply(args);
      any r = result.first();
      if (r is string) return r as string;
      error("'__tostring' must return a string");
    }
  }

  if (a is Table) return "table:" + itos((a as Table).id);
  if (a is UserData) return "userdata:" + itos(((a as UserData) as UserDataInner).id);
  return typestr(a);
}

bool tobool (any a) {
  if (testBool(a)) return getBool(a);
  else if (testNil(a)) return false;
  else return true;
}

bool equals (any a, any b) {
  if (testNil(a) && testNil(b)) return true;
  if (testInt(a) && testInt(b)) return getInt(a) == getInt(b);
  if (testStr(a) && testStr(b)) return getStr(a) == getStr(b);

  float fa, fb; bool t;
  fa, fb, t = getFloats(a, b);
  if (t) return fa == fb;

  if (testBool(a) && testBool(b)) {
    bool _a = getBool(a), _b = getBool(b);
    return (_a && _b) || (!_a && !_b);
  }

  if (testTable(a) && testTable(b)) {
    Table ta = getTable(a);
    Table tb = getTable(b);
    if (ta.id == tb.id) return true;

    any? r = meta_binop(a, b, "__eq");
    if (r.isnull()) return false;
    return tobool(r.get());
  }

  if ((a is UserData) && (b is UserData)) {
    UserDataInner ta = (a as UserData) as UserDataInner;
    UserDataInner tb = (b as UserData) as UserDataInner;
    if (ta.id == tb.id) return true;

    any? r = meta_binop(a, b, "__eq");
    if (r.isnull()) return false;
    return tobool(r.get());
  }

  return false;
}

private any meta_cmp (any a, any b, string key) {
  any? r = meta_binop(a, b, key);
  if (r.isnull())
    error("Lua: attempt to compare " + typestr(a) + " with a " + typestr(b));
  return r.get();
}

private int str_cmp (string a, string b) {
  int al = strlen(a), bl = strlen(b);
  int len = al; if (bl < al) len = bl;
  int i = 0;
  while (i < len) {
    int ca = codeof(charat(a, i));
    int cb = codeof(charat(b, i));
    if (ca < cb) return 0-1;
    if (ca > cb) return 1;
    i = i+1;
  }
  if (al < bl) return 0-1;
  if (al > bl) return 1;
  return 0;
}

private bool _lt (any a, any b) {
  if ((a is int) && (b is int))
    return (a as int) < (b as int);

  float fa, fb; bool t;
  fa, fb, t = getFloats(a, b);
  if (t) return fa < fb;

  if ((a is string) && (b is string))
    return str_cmp(a as string, b as string) < 0;

  return tobool(meta_cmp(a, b, "__lt"));
}

private bool _le (any a, any b) {
  if ((a is int) && (b is int))
    return (a as int) <= (b as int);

  float fa, fb; bool t;
  fa, fb, t = getFloats(a, b);
  if (t) return fa <= fb;

  if ((a is string) && (b is string))
    return str_cmp(a as string, b as string) <= 0;

  return tobool(meta_cmp(a, b, "__le"));
}

any eq (any a, any b) { return equals(a, b) as any; }
any ne (any a, any b) { return anyBool(!equals(a, b)); }
any lt (any a, any b) { return _lt(a, b) as any; }
any le (any a, any b) { return _le(a, b) as any; }
any gt (any a, any b) { return (!_le(a, b)) as any; }
any ge (any a, any b) { return (!_lt(a, b)) as any; }

any not (any a) { return anyBool(!tobool(a)); }
any unm (any a) {
  if (a is int) return (0 - (a as int)) as any;
  if (a is float) return (itof(0) - (a as float)) as any;

  Table? meta = get_metatable(a);
  if (!meta.isnull()) {
    any index = meta.get().get("__unm" as any);
    if (testFn(index)) {
      Function f = getFn(index);
      Stack args = newStack();
      args.push(a);
      Stack result = f.apply(args);
      return result.first();
    }
  }

  error("Lua: attempt to perform arithmetic on a " + typestr(a) + " value");
}

Stack call (any _f, Stack args) {
  if (testFn(_f)) {
    Function f = getFn(_f);
    Stack r = f.apply(args);
    return r;
  } else {
    error("Lua: attempt to call a non-function value");
  }
}


//======= Objects =======//

struct UserDataInner {
  int id;
  any data;
  Table? meta;
}

type UserData (UserDataInner);

UserData newUserData (any data, Table? meta) {
  int id = IdState.id;
  IdState.id = id + 1;
  return (new UserDataInner(id, data, meta)) as UserData;
}


//======= Objects =======//

bool checkKey (any a) { return testStr(a) || testInt(a); }

struct Pair { any key; any val; }

import auro.utils.arraylist (Pair) {
  type `` as PairArr {
    Pair get (int);
    void set (int, Pair);
    int len ();
    void push (Pair);
  }
  PairArr `new` () as emptyPairArr;
}

struct MapPair { string k; any v; }

import auro.utils.stringmap (any) {
  type `` as Map {
    any? get (string);
    void set (string, any);
    void delete (string);
  }
  Map `new` () as newMap;

  type iterator {
    MapPair? next ();
  }
  iterator `new\x1diterator` (Map) as newIter;
}

struct Table {
  int id;

  Map map;
  Array arr;
  PairArr pairs;

  MetaTable? meta;

  iterator? iter;
  string? lastKey;

  // Notes: All keys are valid, only that raw userdata (any type not defined
  // in this module) have no equality, so they can be set but not retrieved.

  // Every key assigned is never removed, only it's value replaced with nil.

  // Int keys greater than the length + 1 are assigned as generic keys, so
  // when the array part catches up, that key will be stored twice.

  // One iterator is mantained, so that for-in loops can run in hopefully
  // constant time, but performance will degrade if next is used arbitrarily.

  // TODO: When auro gets hashmaps, use a single hashmap for everything non-int

  any get (Table this, any key) {
    if (key is int) {
      int k = key as int;
      if ((k > 0) && (k <= this.arr.len()))
        return this.arr[k-1];
    } else if (key is string) {
      any? v = this.map[key as string];
      if (v.isnull()) return nil();
      else return v.get();
    }

    // Fallback
    int i = 0;
    while (i < this.pairs.len()) {
      Pair pair = this.pairs[i];
      if (equals(key, pair.key)) return pair.val;
      i = i+1;
    }
    return nil();
  }

  void set (Table this, any key, any value) {
    if (key is int) {
      int k = key as int;
      if (k == (this.arr.len() + 1)) {
        this.arr.push(value);
      } else  if ((k > 0) && (k <= this.arr.len())) {
        this.arr[k-1] = value;
      } else goto fallback;
    } else if (key is string) {
      this.map[key as string] = value;
    } else {
      fallback:

      int i = 0;
      while (i < this.pairs.len()) {
        Pair pair = this.pairs[i];
        if (equals(key, pair.key)) {
          pair.val = value;
          return;
        }
        i = i+1;
      }
      Pair pair = new Pair(key, value);
      this.pairs.push(pair);
    }

  }

  // This is a complicated function...
  any nextKey (Table this, any key) {

    // If key is nil, just start iterating all keys
    if (testNil(key)) {

      // First integer key
      if (this.arr.len() > 0) return 1 as any;

      first_string_key:
      iterator iter = newIter(this.map);
      MapPair? pair = iter.next();

      // at least one pair, otherwise fallback
      if (!pair.isnull()) {
        string k = pair.get().k;
        this.iter = iter as iterator?;
        this.lastKey = k as string?;
        return k as any;
      }

      first_other_key:
      if (this.pairs.len() > 0)
        return this.pairs[0].key;
      else return nil();
    }

    if (key is int) {
      int k = key as int;
      int len = this.arr.len();
      if ((k > 0) && (k < len)) {
        return (k+1) as any;
      } else if ((k == len) && (len > 0)) {
        // was last integer key
        goto first_string_key;
      } else {
        // outside of array, probably stored as other or not here
        // but definitely not as a string
        goto fallback;
      }
    }

    if (key is string) {
      string k = key as string;

      // Doesn't match current iterator. Find pair that matches
      if (this.lastKey.isnull() || !(this.lastKey.get() == k)) {
        iterator iter = newIter(this.map);
        MapPair? pair = iter.next();

        while (!pair.isnull()) {
          // Found. Save the iterator and proceed
          if (pair.get().k == k) {
            this.iter = iter as iterator?;
            this.lastKey = k as string?;
            goto do_string;
          }
        }

        // Not found, table doesn't have that key
        return nil();
      }

      do_string:
      MapPair? pair = this.iter.get().next();

      if (pair.isnull()) {
        // Was last key, return first callback key or finish
        goto first_other_key;
      } else {
        string k = pair.get().k;
        this.lastKey = k as string?;
        return k as any;
      }
    }

    fallback:
    if (testNil(key) && (this.pairs.len() > 0))
      return this.pairs[0].key;
    int i = 0;
    while (i < this.pairs.len()) {
      Pair pair = this.pairs[i];
      if (equals(key, pair.key)) {
        if ((i+1) < this.pairs.len())
          return this.pairs[i+1].key;
        return nil();
      }
      i = i+1;
    }
    return nil();
  }
}

void table_append (any _t, any _n, Stack stack) {
  Table t = getTable(_t);
  int n = getInt(_n);
  while (stack.more()) {
    t.set(anyInt(n), stack.next());
    n = n+1;
  }
}

type MetaTable (Table);

private Table emptyTable () {
  int id = IdState.id;
  IdState.id = id + 1;
  return new Table(
    id, newMap(), newArray(), emptyPairArr(),
    new MetaTable?(), new iterator?(), new string?()
  );
}
any newTable () { return emptyTable() as any; }

Table? get_metatable (any a) {
  if (testTable(a)) {
    MetaTable? meta = getTable(a).meta;
    if (meta.isnull()) return new Table?();
    return (meta.get() as Table) as Table?;
  }
  if (a is UserData) return ((a as UserData) as UserDataInner).meta;
  if (testStr(a)) return State.string_meta as Table?;
  return new Table?();
}

any get (any a, any k) {
  if (testTable(a)) {
    any val = getTable(a).get(k);
    if (!testNil(val)) return val;
  }
  Table? meta = get_metatable(a);
  if (!meta.isnull()) {
    any index = meta.get().get(anyStr("__index"));
    if (testTable(index)) return get(index, k);
    if (testFn(index)) {
      Function f = getFn(index);
      Stack args = newStack();
      args.push(a);
      args.push(k);
      Stack result = f.apply(args);
      return result.first();
    }
  }
  if (testTable(a)) return nil();
  error("Lua: tried to index a non-table value (" + tostr(a) + ")");
}

void set (any t, any k, any v) {
  if (t is Table) (t as Table).set(k, v);
  else error("Lua: tried to index a non-table value (" + tostr(t) + ")");
}

any length (any a) {
  if (testStr(a)) return anyInt(strlen(getStr(a)));

  Table? mt = get_metatable(a);
  if (!mt.isnull()) {
    any len_fn = mt.get().get(anyStr("__len"));
    if (!testNil(len_fn))
      return call(len_fn, stackof(a)).first();
  }

  if (a is Table) {
    Table t = a as Table;

    // Tentative limit (remember the array is 0-index while lua is 1-index)
    int i = t.arr.len() - 1;

    // TODO: Still needs logarithmic time, using binary search

    if ((i >= 0) && (t.arr[i] is nil_t)) {
      // False limit, must be lower
      while (i >= 0) {
        if (testNil(t.arr[i])) i = i-1;
        else return (i+1) as any;
      }
      return 0 as any;
    } else if (t.get((i+2) as any) is nil_t) {
      return (i+1) as any;
    } else {
      // False limit, must be higher
      i = i+3;
      while (true) {
        if (t.get(anyInt(i)) is nil_t)
          return anyInt(i-1);
        i = i+1;
      }
    }
  }
  error("Lua: attempt to get length of a " + typestr(a) + " value");
}

//======= State =======//

struct IdStateT { int id; }
struct StateT {
  bool ready;
  Table _G;
  Table string;
  Table string_meta;
  Table file_meta;
  Table loaded;
}

StateT State = new StateT(false, emptyTable(), emptyTable(), emptyTable(), emptyTable(), emptyTable());
IdStateT IdState = new IdStateT(0);


//======= Core Library =======//

// Helpers
private string simple_string (any a, string n, string fname) {
  if (testStr(a)) return getStr(a);
  if (testInt(a)) return itos(getInt(a));
  error("Lua: bad argument #" + n + " to '" + fname + "' (string expected, got " + typestr(a) + ")");
}
private int simple_number (any a, string n, string fname) {
  any a = getNum(a);
  if (a is int) return a as int;
  error("Lua: bad argument #" + n + " to '" + fname + "' (number expected, got " + typestr(a) + ")");
}
private int simple_number_or (any a, int d, string n, string fname) {
  if (testNil(a)) return d;
  return simple_number(a, n, fname);
}

private Stack stackof (any a) {
  Stack stack = newStack();
  stack.push(a);
  return stack;
}

Stack _print (Stack args) {
  bool first = true;
  string str = "";
  while (args.more()) {
    any a = args.next();
    if (first) first = false;
    else str = str + "\t";
    str = str + tostr(a);
  }
  println(str);
  return newStack();
} import module newfn (_print) { Function `` () as __print; }

Stack _assert (Stack args) {
  any val = args.next();
  if (tobool(val)) {
    Stack ret = newStack();
    ret.push(val);
    return ret;
  } else {
    any amsg = args.next();
    string msg = tostr(amsg);
    if (testNil(amsg)) msg = "assertion failed!";
    error(msg);
  }
} import module newfn (_assert) { Function `` () as __assert; }

Stack _error (Stack args) { error(tostr(args.next())); }
import module newfn (_error) { Function `` () as __error; }

Stack _tostring (Stack args) { return stackof(anyStr(tostr(args.next()))); }
import module newfn (_tostring) { Function `` () as __tostring; }

Stack _tonumber (Stack args) {
  any a = args.next();
  if ((a is int) || (a is float)) return stackof(a);
  if (a is string) return stackof(parseNum(a as string));
  return newStack();
}
import module newfn (_tonumber) { Function `` () as __tonumber; }

Stack _type (Stack args) { return stackof(anyStr(typestr(args.next()))); }
import module newfn (_type) { Function `` () as __type; }

Stack _getmeta (Stack args) {
  int v = args.next();
  if (testNil(v)) error("Lua: bad argument #1 to 'getmetatable' (value expected)");
  Table? meta = get_metatable(v);
  if (!meta.isnull()) {
    return stackof(anyTable(meta.get()));
  }
  return stackof(nil());
}
import module newfn (_getmeta) { Function `` () as __getmeta; }

Stack _setmeta (Stack args) {
  any a = args.next(), b = args.next();
  if (!testTable(a)) error("Lua: bad argument #1 to 'getmetatable' (table expected, got "+typestr(a)+")");
  if (!testTable(b)) error("Lua: bad argument #2 to 'getmetatable' (table expected, got "+typestr(b)+")");

  Table t = getTable(a);
  Table meta = getTable(b);
  t.meta = (meta as MetaTable) as MetaTable?;

  return stackof(a);
}
import module newfn (_setmeta) { Function `` () as __setmeta; }

Stack _next (Stack args) {
  any a = args.next();
  if (!testTable(a)) error("Lua: bad argument #1 to 'next' (table expected, got "+typestr(a)+")");
  any key = args.next();
  Table t = getTable(a);
  return stackof(t.nextKey(key));
}
import module newfn (_next) { Function `` () as __next; }

Stack _pack (Stack args) {
  Table t = emptyTable();
  int n = 0;
  while (args.more()) {
    n = n+1;
    t.set(anyInt(n), args.next());
  }
  t.set(anyStr("n"), anyInt(n));
  return stackof(anyTable(t));
}
import module newfn (_pack) { Function `` () as __pack; }

Stack _unpack (Stack args) {
  any a = args.next();
  if (!testTable(a)) error("Lua: bad argument #1 to 'table.unpack' (table expected, got "+typestr(a)+")");
  Table t = getTable(a);

  int i = simple_number_or(args.next(), 1, "2", "table.unpack");
  int j = simple_number_or(args.next(), getInt(length(a)), "3", "table.unpack");

  Stack stack = newStack();
  while (i <= j) {
    stack.push(t.get(anyInt(i)));
    i = i+1;
  }
  return stack;
}
import module newfn (_unpack) { Function `` () as __unpack; }

Stack _select (Stack args) {
  any a = args.next();
  if (testStr(a) && (getStr(a) == "#")) return stackof(anyInt(args.length()));
  int index = simple_number(a, "1", "select");
  if (index < 1) error("bad argument #1 to 'select' (index out of range)");
  args.pos = index;
  return args;
}
import module newfn (_select) { Function `` () as __select; }


import lua_lib.table { Stack lua_main (any) as table_main; }


//======= IO and OS =======//

import auro.io {
  type file as File;
  type mode as FileMode;
  FileMode r() as r_mode;
  FileMode w() as w_mode;
  FileMode a() as a_mode;
  File open (string, FileMode);
  buffer read (File, int);
  bool eof (File);
  void write (File, buffer);
  void close (File);
}

File get_file (any a) {
  if (a is UserData) {
    UserDataInner ud = (a as UserData) as UserDataInner;
    if (ud.data is File) {
      return ud.data as File;
    } else error("bad argument #1 to 'read' (file expected)");
  } else error("bad argument #1 to 'read' (file expected, got "+typestr(a)+")");
}

Stack _open (Stack args) {
  string filename = simple_string(args.next(), "1", "io.open");
  any _s = args.next();
  string s = "";

  if (testNil(_s)) s = "r";
  else if (_s is string) s = _s as string;

  FileMode m;
  if      ((s == "r") || (s == "rb")) m = r_mode();
  else if ((s == "w") || (s == "wb")) m = w_mode();
  else if ((s == "a") || (s == "ab")) m = a_mode();
  else error("bad argument #2 to 'io.open' (invalid mode)");

  File file = open(filename, m);
  UserData ud = newUserData(file as any, State.file_meta as Table?);
  return stackof(ud as any);
}
import module newfn (_open) { Function `` () as __open; }

Stack _read (Stack args) {
  File file = get_file(args.next());
  any b = args.next();

  string str;

  if (b is nil_t) b = "l" as any;

  if (b is string) {
    string fmt = b as string;
    if (fmt == "a") {
      str = "";
      repeat:
      buffer buf = read(file, 128);
      str = str + newstr(buf);
      if (!eof(file)) goto repeat;
    } else if (fmt == "n") {
      error("format 'n' not yet supported");
    } else if (fmt == "l") {
      error("format 'l' not yet supported");
    } else if (fmt == "L") {
      error("format 'L' not yet supported");
    } else error("bad argument #2 to 'read' (invalid format)");
  } else if (b is int) {
    str = newstr(read(file, b as int));
  } else error("bad argument #2 to 'read' (invalid format)");

  return stackof(str as any);
}
import module newfn (_read) { Function `` () as __read; }

Stack _write (Stack args) {
  File file = get_file(args.next());

  int i = 1;

  string str = "";
  while (args.more()) {
    str = str + simple_string(args.next(), itos(i), "write");
    i = i+1;
  }

  write(file, tobuffer(str));

  return newStack();
}
import module newfn (_write) { Function `` () as __write; }

Stack _close (Stack args) {
  File file = get_file(args.next());
  close(file);
  return newStack();
}
import module newfn (_close) { Function `` () as __close; }

Stack _filestr (Stack args) {
  any a = args.next();
  File file = get_file(a); // trigger error
  int id = ((a as UserData) as UserDataInner).id;
  string s = "file (" + itos(id) + ")";
  return stackof(s as any);
}
import module newfn (_filestr) { Function `` () as __filestr; }

Stack _exit (Stack args) {
  any a = args.next();
  int code = 0;
  if (a is bool) {
    if (a as bool) code = 0; else code = 1;
  } else {
    code = simple_number_or(a, 0, "1", "os.exit");
  }
  exit(code);
  return newStack();
}
import module newfn (_exit) { Function `` () as __exit; }



//======= String functions =======//

import lua_lib.pattern { Stack lua_main (any) as pattern_main; }
import lua_lib.string { Stack lua_main (any) as string_main; }

int valid_start_index (int i, int len) {
  if (i < 0) i = len+i; else i = i-1;
  if (i < 0) return 0;
  return i;
}

int valid_end_index (int i, int len) {
  if (i < 0) i = len+i; else i = i-1;
  if (i >= len) return len-1;
  return i;
}

Stack _strsubstr (Stack args) {
  string s = simple_string(args.next(), "1", "string.sub");
  int len = strlen(s);

  int i = valid_start_index(simple_number(args.next(), "2", "string.sub"), len);

  int j = len; any _j = args.next();
  if (!testNil(_j)) j = valid_end_index(simple_number(_j, "3", "string.sub"), len);

  string s2 = slice(s, i, j+1);

  return stackof(anyStr(s2));
}
import module newfn (_strsubstr) { Function `` () as __strsubstr; }

Stack _strbyte (Stack args) {
  string s = simple_string(args.next(), "1", "string.byte");
  int len = strlen(s);

  int i = 0; any _i = args.next();
  if (!testNil(_i)) i = valid_start_index(simple_number(_i, "2", "string.byte"), len);
  
  int j = i; any _j = args.next();
  if (!testNil(_j)) j = valid_end_index(simple_number(_j, "2", "string.byte"), len);
  if (j >= len) j = len-1;

  Stack stack = newStack();
  while (i <= j) {
    char ch;
    ch, i = charat(s,i);
    stack.push(anyInt(codeof(ch)));
  }
  return stack;
}
import module newfn (_strbyte) { Function `` () as __strbyte; }

Stack _strchar (Stack args) {
  string s = "";
  int i = 1;
  while (args.more()) {
    int code = simple_number(args.next(), itos(i), "string.char");
    s = addch(s, newchar(code));
    i = i+1;
  }
  return stackof(anyStr(s));
}
import module newfn (_strchar) { Function `` () as __strchar; }



//======= String operations =======//

any, any str_get_nums (Stack args) {
  any a = getNum(args.next());
  any b = getNum(args.next());
  if ((a is nil_t) || (b is nil_t))
    error("attempt to perform arithmetic on a string");
  return a, b;
}

Stack _stradd (Stack args) {
  any a, b; a, b = str_get_nums(args);
  return stackof(add(a, b));
}
import module newfn (_stradd) { Function `` () as __stradd; }

Stack _strsub (Stack args) {
  any a, b; a, b = str_get_nums(args);
  return stackof(sub(a, b));
}
import module newfn (_strsub) { Function `` () as __strsub; }

Stack _strmul (Stack args) {
  any a, b; a, b = str_get_nums(args);
  return stackof(mul(a, b));
}
import module newfn (_strmul) { Function `` () as __strmul; }

Stack _strdiv (Stack args) {
  any a, b; a, b = str_get_nums(args);
  return stackof(div(a, b));
}
import module newfn (_strdiv) { Function `` () as __strdiv; }

Stack _stridiv (Stack args) {
  any a, b; a, b = str_get_nums(args);
  return stackof(idiv(a, b));
}
import module newfn (_stridiv) { Function `` () as __stridiv; }

Stack _strmod (Stack args) {
  any a, b; a, b = str_get_nums(args);
  return stackof(mod(a, b));
}
import module newfn (_strmod) { Function `` () as __strmod; }

Stack _strpow (Stack args) {
  any a, b; a, b = str_get_nums(args);
  return stackof(pow(a, b));
}
import module newfn (_strpow) { Function `` () as __strpow; }

Stack _strunm (Stack args) {
  any a = getNum(args.next());
  if (a is nil_t) error("attempt to perform arithmetic on a string");
  return stackof(unm(a));
}
import module newfn (_strunm) { Function `` () as __strunm; }



//======= Math =======//

Stack _tointeger (Stack args) {
  any a = args.next();
  if (a is string) a = parseNum(a as string);
  if (a is int) return stackof(a);
  if (a is float) {
    float fa = a as float;
    float fr = fa - trunc(fa);
    if (fr == itof(0)) return stackof(ftoi(fa) as any);
  }
  return newStack();
}
import module newfn (_tointeger) { Function `` () as __tointeger; }

Stack _tofloat (Stack args) {
  any a = args.next();
  if (a is string) a = parseNum(a as string);
  if (a is int) return stackof(itof(a as int) as any);
  if (a is float) return stackof(a);
  return newStack();
}
import module newfn (_tofloat) { Function `` () as __tofloat; }

Stack _mathtype (Stack args) {
  any a = args.next();
  if (a is int) return stackof("integer" as any);
  if (a is float) return stackof("float" as any);
  return newStack();
}
import module newfn (_mathtype) { Function `` () as __mathtype; }

import lua_lib.math { Stack lua_main (any) as math_main; }


any get_global () {
  if (!State.ready) {
    // Do not attempt to initialize the state again
    State.ready = true;

    Table tbl = State._G;

    tbl.set(anyStr("_G"), anyTable(tbl));
    tbl.set(anyStr("_VERSION"), anyStr("Lua 5.3"));
    tbl.set(anyStr("_AU_VERSION"), anyStr("0.6"));
    tbl.set(anyStr("assert"), anyFn(__assert()));
    tbl.set(anyStr("error"), anyFn(__error()));
    tbl.set(anyStr("getmetatable"), anyFn(__getmeta()));
    tbl.set(anyStr("next"), anyFn(__next()));
    tbl.set(anyStr("print"), anyFn(__print()));
    // rawequal, rawget, rawlen, rawset
    tbl.set(anyStr("select"), anyFn(__select()));
    tbl.set(anyStr("setmetatable"), anyFn(__setmeta()));
    tbl.set(anyStr("tostring"), anyFn(__tostring()));
    tbl.set(anyStr("tonumber"), anyFn(__tonumber()));
    tbl.set(anyStr("type"), anyFn(__type()));
    // Useless functions:
    // collectgarbage, dofile, load, loadfile

    Table table_tbl = emptyTable();
    tbl.set(anyStr("table"), anyTable(table_tbl));
    table_tbl.set(anyStr("pack"), anyFn(__pack()));
    table_tbl.set(anyStr("unpack"), anyFn(__unpack()));
    table_main(anyTable(State._G));

    State.string_meta.set(anyStr("__index"), anyTable(State.string));
    State.string_meta.set(anyStr("__add"), __stradd() as any);
    State.string_meta.set(anyStr("__sub"), __strsub() as any);
    State.string_meta.set(anyStr("__mul"), __strmul() as any);
    State.string_meta.set(anyStr("__div"), __strdiv() as any);
    State.string_meta.set(anyStr("__unm"), __strunm() as any);
    // idiv mod pow unm (unary minus)

    tbl.set(anyStr("string"), anyTable(State.string));
    State.string.set(anyStr("sub"), anyFn(__strsubstr()));
    State.string.set(anyStr("byte"), anyFn(__strbyte()));
    State.string.set(anyStr("char"), anyFn(__strchar()));
    // These functions can be done in pure lua
    // string.lua: format, len, lower, rep, reverse, upper
    // pattern.lua: find, gmatch, gsub, match
    // pack.lua: pack, packsize, unpack
    string_main(anyTable(State._G));
    pattern_main(anyTable(State._G));

    Table io_tbl = emptyTable();
    tbl.set("io" as any, io_tbl as any);
    io_tbl.set("open" as any, __open() as any);

    State.file_meta.set("__index" as any, State.file_meta as any);
    State.file_meta.set("read" as any, __read() as any);
    State.file_meta.set("write" as any, __write() as any);
    State.file_meta.set("close" as any, __close() as any);
    State.file_meta.set("__tostring" as any, __filestr() as any);

    Table os_tbl = emptyTable();
    tbl.set("os" as any, os_tbl as any);
    os_tbl.set("exit" as any, __exit() as any);

    Table math_tbl = emptyTable();
    tbl.set("math" as any, math_tbl as any);
    math_tbl.set("tointeger" as any, __tointeger() as any);
    math_tbl.set("tofloat" as any, __tofloat() as any);
    math_tbl.set("type" as any, __mathtype() as any);
    math_main(State._G as any);


    // Missing libraries
    // io and os libraries, math
    // the table library can be made in pure lua

    // package is mostly useless in Cobre
    // I'm not sure if i'll implement coroutine, 

    Table arg_tbl = emptyTable();
    int i = 0;
    while (i < argc()) {
      arg_tbl[i as any] = argv(i) as any;
      i = i+1;
    }

    tbl["arg" as any] = arg_tbl as any;
  }
  return anyTable(State._G);
}
